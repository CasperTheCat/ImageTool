#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuRuntime.cuh"
#include <iostream>
#include <stdio.h>
/*#include "hip/hip_vector_types.h"
#include "helper_image.h"*/
/*#include "hip/hip_fp16.h"
#include "cudaGL.h"
#include "cudaD3D11.h"
#include "cuda_d3d11_interop.h"*/
/*#include "math_functions.h"*/

#define LUMA float3{float(0.2126), float(0.7152), float(0.0722)}

// Define LUMA
//float3 LUMA{ float(0.2126), float(0.7152), float(0.0722) };


__device__ inline float dot(float3 iX, float3 iY)
{
	return (iX.x * iY.x) + (iX.y * iY.y) + (iX.z * iY.z);
}

__device__ inline float dot(float iX, float3 iY)
{
	return (iX * iY.x) + (iX * iY.y) + (iX * iY.z);
}

__device__ inline float dot(float4 iX, float3 iY)
{
	return (iX.x * iY.x) + (iX.y * iY.y) + (iX.z * iY.z);
}



// REM hipFree
#pragma region Define Operators
__device__
float3 operator- (float3 f1, float f2)
{
	return float3{ f1.x - f2, f1.y - f2, f1.z - f2 };
}

__device__
float3 operator+ (float3 f1, float f2)
{
	return float3{ f1.x + f2, f1.y + f2, f1.z + f2 };
}

__device__
float3 operator/ (float3 f1, float f2)
{
	return float3{ f1.x / f2, f1.y / f2, f1.z / f2 };
}

__device__
float3 operator* (float3 f1, float f2)
{
	return float3{ f1.x * f2, f1.y * f2, f1.z * f2 };
}


#pragma endregion
__host__ __device__
int Modulus(int a, int b)
{
	int ret = a % b;
	if (ret < 0)
		ret += b;
	return ret;
}

template <typename T>
__host__ __device__
inline T lerp(T v0, T v1, T t)
{
	return fma(t, v1, fma(-t, v0, v0));
}

__device__ inline float gpuClamp(float in_t, int high_i, int low_i)
{
	return (in_t < low_i) ? low_i : (in_t > high_i) ? high_i : in_t;
}

__device__ inline float gpuClamp(float in_t)
{
	return (in_t < 0.f) ? 0.f : (in_t > 1.f) ? 1.f : in_t;
}

__device__ inline float CTF(unsigned char in)
{
	return (float(in) / float(255));
}

__device__ inline unsigned char FTC(float in)
{
	return unsigned char(gpuClamp(in) * 255);
}

// PreProcessor, does a transpose op
__global__ void kPreProcess(unsigned char* data, float3* out, uint64_t len)
{
	uint64_t gIdx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t dataLocation = gIdx * 4;
	if (dataLocation >= len) return;

	out[gIdx].x = CTF(data[dataLocation]);
	out[gIdx].y = CTF(data[dataLocation + 1]);
	out[gIdx].z = CTF(data[dataLocation + 2]);

}

// PostProcessor, does a reverse transpose op
__global__ void kPostProcess(unsigned char* data, float3* in, uint64_t len)
{
	uint64_t gIdx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t dataLocation = gIdx * 4;
	if (dataLocation >= len) return;

	data[dataLocation] = FTC(in[gIdx].x);
	data[dataLocation + 1] = FTC(in[gIdx].y);
	data[dataLocation + 2] = FTC(in[gIdx].z);
	//data[dataLocation + 3] = FTC(1.f);
	// ignore alpha. We left it in the array
}

__device__ float lumaSharp(float3* in, uint64_t len, uint64_t idx, uint32_t pX, uint32_t pY, int fW)
{
	float difUp = dot(in[Modulus(idx - pX * fW, pX * pY)], LUMA);
	float difDown = dot(in[Modulus(idx + pX * fW, pX * pY)], LUMA);
	float difLeft = dot(in[Modulus(idx - 1 * fW, pX * pY)], LUMA);
	float difRight = dot(in[Modulus(idx + 1 * fW, pX * pY)], LUMA);

	// Non Immed
	float difUpRight = dot(in[Modulus((idx - pX * fW) + 1 * fW, pX * pY)], LUMA);
	float difDownLeft = dot(in[Modulus((idx + pX * fW) - 1 * fW, pX * pY)], LUMA);
	float difUpLeft = dot(in[Modulus((idx - pX * fW) - 1 * fW, pX * pY)], LUMA);
	float difDownRight = dot(in[Modulus((idx + pX * fW) + 1 * fW, pX * pY)], LUMA);


	//__syncthreads();
	return (dot(in[idx], LUMA) * 0.12f
		+ difUp * 0.11f
		+ difDown * 0.11f
		+ difLeft * 0.11f
		+ difRight * 0.11f
		+ difUpRight * 0.11f
		+ difUpLeft * 0.11f
		+ difDownLeft * 0.11f
		+ difDownRight * 0.11f);

	/*return (difUp * 0.125f
	+ difDown * 0.125f
	+ difLeft *  0.125f
	+ difRight *  0.125f
	+ difUpRight *  0.125f
	+ difUpLeft *  0.125f
	+ difDownLeft *  0.125f
	+ difDownRight *  0.125f);*/
}


// Main Processor step
__global__ void kProcessStepOne(float3* in, float3* cOut, float* lOut, uint64_t len, uint32_t pX, uint32_t pY)
{
	uint64_t gIdx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t dataLocation = gIdx * 4;
	if (gIdx >= pY*pX) { return; }

	//float3 temp = in[gIdx];
	// Strip Luma!
	float pLuma = dot(in[gIdx], LUMA);
	float3 pChro = in[gIdx] - pLuma;



	// Sharpen - Get Blur
	float blur = lumaSharp(in, len, gIdx, pX, pY,1);


	// Sharpen - Sub Blur from Luma
	float sharp = pLuma - blur;




	// Bloom?

	// Edge Detect


	// Contrast
	pChro = ((pChro - 0.5f) * 1.1f) + 0.5f;

	// % Brightness
	float vLen = dot(1.0, pChro) / 3;
	pChro = pChro * (1.25f + vLen);
	//pChro = pChro * 1.5f;

	cOut[gIdx] = pChro;
	lOut[gIdx] = pLuma + dot(sharp, LUMA);

	/*__syncthreads();
	in[gIdx] = pChro + pLuma + dot(sharp, LUMA * 2);*/
	//in[gIdx] = { blur,blur,blur };
}

// Stitch Buffers
__global__ void kProcessStepTwo(float3* out, float3* cIn, float* lIn, uint64_t len, uint32_t pX, uint32_t pY)
{
	uint64_t gIdx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t dataLocation = gIdx * 4;
	if (gIdx >= pY*pX) { return; }

	// Stitch data
	out[gIdx] = cIn[gIdx] + lIn[gIdx];
	//in[gIdx] =  + pLuma + dot(sharp, LUMA * 2);
	//in[gIdx] = { blur,blur,blur };
}


bool _GPUSelected =false;
void gpuSelector(unsigned char* data, uint64_t len, uint32_t pX, uint32_t pY)
{
	// Get Device
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (!_GPUSelected)
	{
		if (deviceCount == 0)
		{
			std::cout << "CUDA Capable GPU was not detected" << std::endl;
			return;
		}
		else
		{
			std::cout << deviceCount << " CUDA capable GPUs were detected" << std::endl;
			_GPUSelected = true;
		}
	}

	if (deviceCount > 1) 
	{
		hipSetDevice(0);
		std::cout << "Using Device 1" << std::endl;
	}
	else
	{
		hipSetDevice(0);
		std::cout << "Using Device 0" << std::endl;
	}

	// Threading
	uint32_t threadCount = 1024;
	uint32_t blockCount = (pY * pX) / threadCount + 1;

	// Setup the transpose!
	float3* imageData;
	hipMalloc((void**)&imageData, sizeof(float3) * pX * pY);
	float3* chromaData;
	hipMalloc((void**)&chromaData, sizeof(float3) * pX * pY);
	float* lumaData;
	hipMalloc((void**)&lumaData, sizeof(float) * pX * pY);

	std::cout << "cMalloc: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// Run Transpose
	kPreProcess << < blockCount, threadCount >> > (data, imageData, len);
	hipDeviceSynchronize();
	std::cout << "PreProcess: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// Run Process
	kProcessStepOne << < blockCount, threadCount >> >(imageData, chromaData, lumaData, len, pX, pY);
	hipDeviceSynchronize();
	std::cout << "Process: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// Sync the GPU. We need step 1 to finish before step is scheduled
	hipDeviceSynchronize();
	std::cout << "Synchronize: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// Run Stitch
	kProcessStepTwo << < blockCount, threadCount >> >(imageData, chromaData, lumaData, len, pX, pY);
	std::cout << "Stitch " << hipGetErrorString(hipGetLastError()) << std::endl;

	hipFree(chromaData);
	hipFree(lumaData);
	std::cout << "Free: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// Run antiTranspose
	kPostProcess << < blockCount, threadCount >> > (data, imageData, len);


	// Sync
	hipDeviceSynchronize();
	std::cout << "PostProcess: " << hipGetErrorString(hipGetLastError()) << std::endl;

	// free
	hipFree(imageData);
}


void gpuProcess(unsigned char* data, uint64_t dataLength, uint32_t pX, uint32_t pY)
{
	// MEMCPY to GPU
	unsigned char* deviceData;
	hipMalloc((void**)&deviceData, sizeof(unsigned char) * dataLength);


	hipMemcpy(deviceData, data, dataLength, hipMemcpyHostToDevice);


	gpuSelector(deviceData, dataLength, pX, pY);



	hipMemcpy(data, deviceData, dataLength, hipMemcpyDeviceToHost);

	hipFree(deviceData);

	// Make sure GPU is in a good state!
	hipDeviceReset();
}
