#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuRuntime.cuh"
#include <iostream>
#include <stdio.h>


// OPTIONS
// These are options, they will be made later

// Darkness
#define darkPivot -0.1f // DEF 0.1f

// uSharp
#define uSharpStrength 1.f
#define uSharpWidth 1

// Contrast
#define contraStrength 1.1f
#define contraPivot 0.5f

// Saturation
#define satStrength 1.5f

// Experimental
#define r_experimental 0


// END OPTIONS

#define LUMA float3{float(0.2126), float(0.7152), float(0.0722)}

// Define LUMA
//float3 LUMA{ float(0.2126), float(0.7152), float(0.0722) };


__device__ inline float dot(float3 iX, float3 iY)
{
	return (iX.x * iY.x) + (iX.y * iY.y) + (iX.z * iY.z);
}

__device__ inline float dot(float iX, float3 iY)
{
	return (iX * iY.x) + (iX * iY.y) + (iX * iY.z);
}

__device__ inline float dot(float4 iX, float3 iY)
{
	return (iX.x * iY.x) + (iX.y * iY.y) + (iX.z * iY.z);
}


#pragma region Define Operators F4
__device__
float4 operator- (float4 f1, float f2)
{
	return float4{ f1.x - f2, f1.y - f2, f1.z - f2, f1.w };
}

__device__
float4 operator- (float4 f1, float4 f2)
{
	return float4{ f1.x - f2.x, f1.y - f2.y, f1.z - f2.z, f1.w };
}

__device__
float4 operator+ (float4 f1, float f2)
{
	return float4{ f1.x + f2, f1.y + f2, f1.z + f2, f1.w };
}

__device__
float4 operator+ (float4 f1, float4 f2)
{
	return float4{ f1.x + f2.x, f1.y + f2.y, f1.z + f2.z, f1.w };
}

__device__
float4 operator/ (float4 f1, float f2)
{
	return float4{ f1.x / f2, f1.y / f2, f1.z / f2, f1.w };
}

__device__
float4 operator/ (float f1, float4 f2)
{
	return float4{ f1 / f2.x, f1 / f2.y, f1 / f2.z, f2.w };
}

__device__
float4 operator* (float4 f1, float f2)
{
	return float4{ f1.x * f2, f1.y * f2, f1.z * f2, f1.w };
}

__device__
float4 operator* (float4 f1, float4 f2)
{
	return float4{ f1.x * f2.x, f1.y * f2.y, f1.z * f2.z, f1.w };
}


#pragma endregion

#pragma region Define Operators
__device__
float3 operator- (float3 f1, float f2)
{
	return float3{ f1.x - f2, f1.y - f2, f1.z - f2 };
}

__device__
float3 operator- (float3 f1, float3 f2)
{
	return float3{ f1.x - f2.x, f1.y - f2.y, f1.z - f2.z };
}

__device__
float3 operator+ (float3 f1, float f2)
{
	return float3{ f1.x + f2, f1.y + f2, f1.z + f2 };
}

__device__
float3 operator+ (float3 f1, float3 f2)
{
	return float3{ f1.x + f2.x, f1.y + f2.y, f1.z + f2.z };
}

__device__
float3 operator/ (float3 f1, float f2)
{
	return float3{ f1.x / f2, f1.y / f2, f1.z / f2 };
}

__device__
float3 operator/ (float f1, float3 f2)
{
	return float3{ f1 / f2.x, f1 / f2.y, f1 / f2.z };
}

__device__
float3 operator* (float3 f1, float f2)
{
	return float3{ f1.x * f2, f1.y * f2, f1.z * f2 };
}

__device__
float3 operator* (float3 f1, float3 f2)
{
	return float3{ f1.x * f2.x, f1.y * f2.y, f1.z * f2.z };
}


#pragma endregion

#pragma region Helpers
__host__ __device__
int Modulus(int a, int b)
{
	int ret = a % b;
	if (ret < 0)
		ret += b;
	return ret;
}

template <typename T>
__host__ __device__
inline T lerp(T v0, T v1, T t)
{
	return fma(t, v1, fma(-t, v0, v0));
}

__device__ inline float3 lerp(float3 fX, float3 fY, float fZ)
{
	return fX*(1 - fZ) + fY*fZ;
}
__device__ inline float4 lerp(float4 fX, float4 fY, float fZ)
{
	return fX*(1 - fZ) + fY*fZ;
}

__device__ inline float gpuClamp(float in_t, int high_i, int low_i)
{
	return (in_t < low_i) ? low_i : (in_t > high_i) ? high_i : in_t;
}

__device__ inline float gpuClamp(float in_t)
{
	return (in_t < 0.f) ? 0.f : (in_t > 1.f) ? 1.f : in_t;
}

__device__ inline float CTF(unsigned char in)
{
	return (float(in) / float(255));
}

__device__ inline unsigned char FTC(float in)
{
	return unsigned char(gpuClamp(in) * 255);
}

#pragma endregion


// PreProcessor, does a transpose op
__global__ void kPreProcess(unsigned char* data, float4* out, uint64_t len)
{
	uint64_t gIdx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t dataLocation = gIdx * 4;
	if (dataLocation >= len) return;

	out[gIdx].x = CTF(data[dataLocation]);      // R
	out[gIdx].y = CTF(data[dataLocation + 1]);  // G
	out[gIdx].z = CTF(data[dataLocation + 2]);  // B
	out[gIdx].w = CTF(data[dataLocation + 3]);  // A
}




// PostProcessor, does a reverse transpose op
__global__ void kPostProcess(unsigned char* data, float4* in, uint64_t len)
{
	uint64_t gIdx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t dataLocation = gIdx * 4;
	if (dataLocation >= len) return;

	data[dataLocation] = FTC(in[gIdx].x);
	data[dataLocation + 1] = FTC(in[gIdx].y);
	data[dataLocation + 2] = FTC(in[gIdx].z);
	data[dataLocation + 3] = FTC(in[gIdx].w);
}


// Bloom Effect
// Experimental Bloom
//
__device__ float4 r_bloom(float4* aColor, uint64_t idx)
{
	// Not Yet Implemented
	return aColor[idx] * aColor[idx];
}


// Device lumaSharpen
// Sharpen the image with a uSharp Mask
//
__device__ float lumaSharp(float4* in, uint64_t idx, uint32_t pX, uint32_t pY, int fW)
{
	float difUp = dot(in[Modulus(idx - pX * fW, pX * pY)], LUMA);
	float difDown = dot(in[Modulus(idx + pX * fW, pX * pY)], LUMA);
	float difLeft = dot(in[Modulus(idx - 1 * fW, pX * pY)], LUMA);
	float difRight = dot(in[Modulus(idx + 1 * fW, pX * pY)], LUMA);

	// Non Immed
	float difUpRight = dot(in[Modulus((idx - pX * fW) + 1 * fW, pX * pY)], LUMA);
	float difDownLeft = dot(in[Modulus((idx + pX * fW) - 1 * fW, pX * pY)], LUMA);
	float difUpLeft = dot(in[Modulus((idx - pX * fW) - 1 * fW, pX * pY)], LUMA);
	float difDownRight = dot(in[Modulus((idx + pX * fW) + 1 * fW, pX * pY)], LUMA);


	//__syncthreads();
	return (dot(in[idx], LUMA) * 0.12f
		+ difUp * 0.11f
		+ difDown * 0.11f
		+ difLeft * 0.11f
		+ difRight * 0.11f
		+ difUpRight * 0.11f
		+ difUpLeft * 0.11f
		+ difDownLeft * 0.11f
		+ difDownRight * 0.11f);
}



// Main Processor step
__global__ void kProcess(float4* cIn, float4* cOut, uint32_t pX, uint32_t pY)
{
	uint64_t gIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (gIdx >= pY*pX) { return; }

	// Strip Luma!
	float pLuma = dot(cIn[gIdx], LUMA);
	float4 pChro = cIn[gIdx] - pLuma;


	// pDarkness. Pivot / Color. 
	// Add in Central Shift
	// Mid Shift = 2Pivot, from Pivot / 0.5f 
	if (darkPivot > 0.f)
	{
		float pDark = (darkPivot / pLuma) - (darkPivot * 2);
		pLuma -= pDark;
	}

	// Sharpen
	// This effect is similar to the Adobe UnSharp Mask
	float sharpMask = 0.f; // Default to Zero
	if (uSharpWidth > 0.f && uSharpStrength > 0.f)
	{
		// Sharpen - Get Blur
		float blur = lumaSharp(cIn, gIdx, pX, pY, uSharpWidth);
		// Sharpen - Sub Blur from Luma
		sharpMask = dot(pLuma - blur, LUMA * uSharpStrength);
	}

	// Bloom?
#if r_experimental
	pChro = r_bloom(in,gIdx);
#endif


	// Aberration
	//float vLen = dot(1.0, pChro) / 3;
	//pChro = pChro * (1.25f + vLen);
	// or
	//pChro = pChro * ( 1.f + (dot(1.0, pChro) / 3));

	// Saturate and restitch
	float4 cStitch = pChro + (pLuma + sharpMask);
	float4 sColor = lerp(float4{ (pLuma + sharpMask), (pLuma + sharpMask), (pLuma + sharpMask), pChro.w }, cStitch, satStrength);

	// Contrast and save to array
	cOut[gIdx] = ((sColor - contraPivot) * contraStrength) + contraPivot;
	//cOut[gIdx] = pChro +pLuma + sharpMask;
}

void gpuSelector(unsigned char* data, uint64_t len, uint32_t pX, uint32_t pY, int fGPU)
{
	// Useless call. Extremely low overhead so I'm remove and validate later
    hipSetDevice(fGPU);
	
	// Threading values
	uint32_t threadCount = 1024;
	uint32_t blockCount = (pY * pX) / threadCount + 1;

	// Image data is Data from the CPU
	float4* inImageData;
	hipMalloc((void**)&inImageData, sizeof(float4) * pX * pY);

	// Where finished data will go
	float4* outImageData;
	hipMalloc((void**)&outImageData, sizeof(float4) * pX * pY);

	// Run Transpose - faster on GPU until I write the Intel SIMD stuff
	kPreProcess <<< blockCount, threadCount >>> (data, inImageData, len);
	
	// Sync the GPU. This is barrier
	hipDeviceSynchronize();
	
	// Run Main
	kProcess <<< blockCount, threadCount >>>(inImageData, outImageData, pX, pY);
	
	// Sync the GPU. Another Barrier
	hipDeviceSynchronize();

	// We are done with the initial data from the CPU
	hipFree(inImageData);

	// Revert the formatting of the data
	kPostProcess <<< blockCount, threadCount >>> (data, outImageData, len);

	// Sync before final free
	hipDeviceSynchronize();
	hipFree(outImageData);
}


void gpuProcess(unsigned char* data, uint64_t dataLength, uint32_t pX, uint32_t pY, int iGPU)
{
        // Make sure the memory is allocated on the correct GPU 
        hipSetDevice(iGPU);

	// MEMCPY to GPU
	unsigned char* deviceData;
	hipMalloc((void**)&deviceData, sizeof(unsigned char) * dataLength);


	hipMemcpy(deviceData, data, dataLength, hipMemcpyHostToDevice);


	gpuSelector(deviceData, dataLength, pX, pY, iGPU);



	hipMemcpy(data, deviceData, dataLength, hipMemcpyDeviceToHost);

	hipFree(deviceData);

	// Make sure GPU is in a good state!
	hipDeviceReset();
}



// GPU COUNT
// Returns GPU Count

int gpuCount()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	return deviceCount;
}
